#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>

#ifndef ARRAY_SIZE
#define ARRAY_SIZE 10000
#endif

// Theards per block
#ifndef TPB
#define TPB 256
#endif

// DEBUG
#ifndef PRINT
#define PRINT 0
#endif

// Returns a random float between 0 and 1
float rand_float();

// Initialize a random vector of size ARRAY_SIZE
void init_rand_vec(float*);

// Serial CPU SAXPY code
void saxpy_serial_cpu(float, float*, float*);

// CUDA GPU kernel for SAXPY
__global__ void saxpyKernel(float, float*, float*);

// Compute the sum of squares mean error
float l2_avg_norm(float*, float*);

// Timing
double cpuSecond();

// For debug
void print_vector(float*);

int main()
{

	// Print array size
	printf("\nArray size = %d\n", ARRAY_SIZE);

	// Init RNG
	srand(time(0));

	float x[ARRAY_SIZE];
	float y1[ARRAY_SIZE];
	float a = rand_float();
	init_rand_vec(x);
	init_rand_vec(y1);

	if (PRINT)
	{
        	printf("\n*** Debug: print CPU init ***\n");
        	print_vector(y1);
	}

	// Init GPU x and y vectors
	float* x_gpu=NULL;	
	float* y_gpu=NULL;
	hipMalloc(&x_gpu, ARRAY_SIZE*sizeof(float));
	hipMalloc(&y_gpu, ARRAY_SIZE*sizeof(float));
	hipMemcpy(x_gpu, x, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y_gpu, y1, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);

	// Perform SAXPY serially on CPU
	double cpu_time_start = cpuSecond();
	saxpy_serial_cpu(a, x, y1);
	double cpu_time_elaps = cpuSecond() - cpu_time_start;
	printf("\nComputing SAXPY on the CPU… Done!\nTime CPU = %fsec\n", cpu_time_elaps);
	
	if (PRINT)
	{
		printf("\n*** Debug: print CPU result ***\n");
		print_vector(y1);
	}

	// Perform SAXPY on GPU
	double gpu_time_start = cpuSecond();
	saxpyKernel<<<(ARRAY_SIZE+TPB-1)/TPB, TPB>>>(a, x_gpu, y_gpu);
	hipDeviceSynchronize();
	double gpu_time_elaps = cpuSecond() - gpu_time_start;
	printf("\nComputing SAXPY on the GPU… Done!\nTime GPU = %fsec\n", gpu_time_elaps);

	// Copy back to CPU
	float y2[ARRAY_SIZE];
	hipMemcpy(y2, y_gpu, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);

	if (PRINT)
        {
        	printf("\n*** Debug: print GPU result ***\n");
        	print_vector(y2);
	}

	hipFree(x_gpu);
	hipFree(y_gpu);
	
	float l2_error = l2_avg_norm(y1, y2);
	printf("\nL2 norm error = %f\n", l2_error);

	return 1;

}

float rand_float()
{
	return ((float)rand()) / RAND_MAX;
}

void init_rand_vec(float* vec)
{
	for (int n = 0; n<ARRAY_SIZE; n++)
		vec[n] = rand_float();
}

void saxpy_serial_cpu(float a, float* x, float* y)
{
	for (int n=0; n<ARRAY_SIZE; n++)
		y[n] += a*x[n];
}

__global__ void saxpyKernel(float a, float* x, float* y)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	y[i] += a*x[i];
	/* Test */
	// y[i] = i;
	/* **** */
}

float l2_avg_norm(float* v1, float* v2)
{
	float sum = 0.0;
	float diff;
	for (int n = 0; n < ARRAY_SIZE; n++) 
	{	
		diff = v1[n]-v2[n];
		sum += diff*diff;
	}
	return sqrt(sum)/ARRAY_SIZE;
}

double cpuSecond() 
{
  	struct timeval tp;
   	gettimeofday(&tp,NULL);
   	return ((double)tp.tv_sec + (double)tp.tv_usec*1.0e-6);
}

void print_vector(float* x)
{
	for (int n = 0; n<ARRAY_SIZE; ++n)
		printf("%f\n", x[n]);
}
