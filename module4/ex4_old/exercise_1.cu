
#include <hip/hip_runtime.h>
#include <stdio.h>

// Theards per block
#ifndef TPB
#define TPB 256
#endif

// Blocks
#ifndef NB
#define NB 1
#endif

__global__ void helloWorldKernel();

int main()
{
	
	// Launch kernel
	helloWorldKernel<<<NB, TPB>>>();

	// Syncronize
	hipDeviceSynchronize();

	return 0;

}

__global__ void helloWorldKernel()
{
	const int th_idx = blockIdx.x*blockDim.x + threadIdx.x;
	printf("Hello World! My threadId is %d\n", th_idx);
}
