
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cudaHello()
{
    int const threadId = blockIdx.x * blockDim.x + threadIdx.x;
   
    printf("Hello World! My threadId is %d\n", threadId);
}

int main()
{
    int const TB = 1;       // Number of thread blocks
    int const TPB = 256;    // Number of threads per block

    cudaHello<<<TB, TPB>>>();
    hipDeviceSynchronize();
}
